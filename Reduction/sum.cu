#include "hip/hip_runtime.h"
#include "reduce.h"


__device__ float update(float old,float opOutput,float *extraParams) {
	return opOutput + old;
}


__device__ float merge(float old,float opOutput,float *extraParams) {
	return opOutput + old;
}

__device__ float op(float d1,float *extraParams) {
	return d1;
}


__device__ float postProcess(float reduction,int n,int xOffset,float *dx,int incx,float *params,float *result) {
	return reduction;
}


extern "C"
__global__ void sum_strided_float(int n, int xOffset,float *dx,int incx,float *params,float *result) {
	transform(n,xOffset,dx,incx,params,result);
}



int main(void) {
	void *d = NULL;
	int i;
	int numElements = 12;
	size_t size = numElements * sizeof(float);
	printf("[Vector addition of %d elements]\n", numElements);

	// Allocate the host input vector A
	float *h_A = (float *) malloc(size);
	for(i = 0; i < numElements; i++) {
		h_A[i] = i + 1;
		printf("Host %d \n",i);
	}

    float *h_Result = (float*) malloc(size);
    for(int i = 0; i < numElements; i++) {
    	h_Result[i] = 0;
    }


	float *d_A = NULL;
	//allocate memory on device
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_A, size));
	//copy memory from host to device
	CUDA_CHECK_RETURN(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

	float *d_Result = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_Result, size));
	CUDA_CHECK_RETURN(hipMemcpy(d_Result, h_A, size, hipMemcpyHostToDevice));



	//dot<<<blocksPerGrid,threadsPerBlock>>>( dev_a, dev_b,dev_partial_c );
	int blocksPerGrid = 128;
	int threadsPerBlock = 256;

	float *extraParams = (float *) malloc(1 * sizeof(float));
	extraParams[0] = 0;
	float *d_extraParams = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_extraParams, size));
    CUDA_CHECK_RETURN(hipMemcpy(d_extraParams, extraParams, 1 * sizeof(float), hipMemcpyHostToDevice));


	sum_strided_float<<<blocksPerGrid,threadsPerBlock,512 * sizeof(float)>>>(numElements,0,d_A,1,d_extraParams,d_Result);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy(h_Result, d_Result, size, hipMemcpyDeviceToHost));


	printf("Sum %f\n", h_Result[0]);

	CUDA_CHECK_RETURN(hipFree(d_A));
	CUDA_CHECK_RETURN(hipFree(d_Result));

	free(h_A);
	free(h_Result);
	CUDA_CHECK_RETURN(hipDeviceReset());
	CUDA_CHECK_RETURN(hipFree((void*) d));


	return 0;
}
